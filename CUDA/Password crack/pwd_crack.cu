#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o pwd_crack pwd_crack.cu


     To Run:
     ./pwd_crack > results.txt

  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is
  identical to the plain text password string stored in the program.
  Otherwise,it returns 0.
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char password1[] = "CV78";
  char password2[] = "ES81";
  char password3[] = "GT34";
  char password4[] = "RD48";

  char *a = attempt;
  char *b = attempt;
  char *c = attempt;
  char *d = attempt;
  char *pswd1 = password1;
  char *pswd2 = password2;
  char *pswd3 = password3;
  char *pswd4 = password4;

  while(*a == *pswd1) {
   if(*a== '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    a++;
    pswd1++;
  }
    
  while(*b == *pswd2) {
   if(*b == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    b++;
    pswd2++;
  }

  while(*c == *pswd3) {
   if(*c == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    c++;
    pswd3++;
  }

  while(*d == *pswd4) {
   if(*d == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    d++;
    pswd4++;
  }
  return 0;

}
/****************************************************************************
  The kernel function assume that there will be only one thread and uses
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char p,s;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(p='0'; p<='9'; p++){
      for(s='0'; s<='9'; s++){
            password[2] = p;
            password[3] = s;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}




