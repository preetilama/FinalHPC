#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <crypt.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


/******************************************************************************
  Demonstrates how to crack an encrypted password using a simple
  "brute force" algorithm. Works on passwords that consist only of 2 uppercase
  letters and a 2 digit integer. Your personalised data set is included in the
  code. 

  Compile with:
    nvcc -o cuda_password cuda_password.cu -lcrypt

  If you want to analyse the results then use the redirection operator to send
  output to a file that you can view using an editor or the less utility:

    ./CrackAZ99-With-Data > results.txt

  Dr Kevan Buckley, University of Wolverhampton, 2018
******************************************************************************/


__device__ int is_a_match(char *attempt){
	
char p_password1[] ="CV7812";
char p_password2[] ="ES8122";
char p_password3[] ="GT3433";
char p_password4[] ="RD4844";

	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;

	char *p1 = p_password1;
	char *p2 = p_password2;
	char *p3 = p_password3;
	char *p4 = p_password4;


while (*a == *p1){
	if(*a == '\0'){
		//printf("Password: %s\n", p_password1);
		break;
	}
	a++;
	p1++;
}

while (*b == *p2){
	if(*b == '\0'){
		//printf("Password: %s\n", p_password2);
		break;
	}
	b++;
	p2++;
}

while (*c == *p3){
	if(*c == '\0'){
		//printf("Password: %s\n", p_password3);
		break;
	}
	c++;
	p3++;
}

while (* d== *p4){
	if(*d == '\0'){
		printf("Password: %s\n", p_password4);
		break;
	}
	d++;
	p4++;
}
return 0;

}

__global__ void kernel(){
	char k1,k2,k3,k4;
	
	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;

	char matchone = i;
	char matchtwo = j;
	password[0] = matchone;
	password[1] = matchtwo;
	
	for(k1='0';k1<='9';k1++){
		for(k2='0';k2<='9';k2++){
			for(k3='0';k3<='9';k3++){
				for(k4='0';k4<='9';k4++){
					password[2] = k1;
					password[3] = k2;
					password[4] = k3;
					password[5] = k4;
					if(is_a_match(password)){
											
					}else{
						
					}
					
	}
	}
	}
	}

}



int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}



int main(int argc, char *argv[]){
  struct timespec start, finish;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

 	kernel <<<26,26>>>();
	hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
         (time_elapsed/1.0e9)); 

  return 0;
}
